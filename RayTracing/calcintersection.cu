#include "hip/hip_runtime.h"
#include <qdebug.h>
#include <qelapsedtimer.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "structs.h"
#include "intersectionwizard.h"

/*float *objData = nullptr;
float* dev_objData;
unsigned int objDataBytes = 0;
unsigned int objDataSize = 0;*/

bool temp = false;

static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__device__ float dot_product(float *a, float *b)
{
	return (a[0] * b[0] + a[1] * b[1] + a[2] * b[2]);
}

__device__ void cross_product(float *a, float *b, float *dest)
{
	dest[0] = a[1] * b[2] - a[2] * b[1];
	dest[1] = a[2] * b[0] - a[0] * b[2];
	dest[2] = a[0] * b[1] - a[1] * b[0];
}

__global__ void calcIntersection(float *objData, float *dirData, float *origData, float *outPointsData, float *outLengthsData,
								 unsigned int objDataSize)
{
	int tid =  blockIdx.x * blockDim.x + threadIdx.x; //ID ��������� �� ���� ��� (���� �����������) �� *dirData

	/*������ � dirData:
	0 = float x, 1 = float y, 2 = float z
	3 = float x, 4 = float y, 5 = float z � �.� �� (dirDataSize / sizeof(float)) */

	float dir[3];
	dir[0] = dirData[tid * 3 + 0];
	dir[1] = dirData[tid * 3 + 1];
	dir[2] = dirData[tid * 3 + 2];

	float orig[3];
	orig[0] = origData[0];
	orig[1] = origData[1];
	orig[2] = origData[2];

	outPointsData[tid * 3 + 0] = 0;
	outPointsData[tid * 3 + 1] = 0;
	outPointsData[tid * 3 + 2] = 0;

	float minLength = 50.0;
	outLengthsData[tid] = minLength;

	/*������ � objData:
	0, 1, 2 - ������� 1� �����, 3, 4, 5 - ���������� 1� �����
	6, 7, 8 - ������� 2� �����, 9, 10, 11 - ���������� 2� �����
	12, 13, 14 - ������� 3� �����, 15, 16, 17 - ���������� 3� �����
	� �.� �� (objDataSize / sizeof(float)) */
	for (int i = 0; i < objDataSize; i+=18) //18 - ������ ������ TriangleData
	{
		float epsilon = 0.000001;
		float N[3], v0[3], v1[3], v2[3];

		N[0] = objData[i + 0];
		N[1] = objData[i + 1];
		N[2] = objData[i + 2];

		v0[0] = objData[i + 3];  v1[0] = objData[i + 9];   v2[0] = objData[i + 15];
		v0[1] = objData[i + 4];  v1[1] = objData[i + 10];  v2[1] = objData[i + 16];
		v0[2] = objData[i + 5];  v1[2] = objData[i + 11];  v2[2] = objData[i + 17];

		float denominator = dot_product(dir, N);
		if ((denominator >= 0.0 && denominator < epsilon) ||
			(denominator <  0.0 && denominator > -epsilon))
			continue;

		float v0minOrig[3]; 
		v0minOrig[0] = v0[0] - orig[0];
		v0minOrig[1] = v0[1] - orig[1];
		v0minOrig[2] = v0[2] - orig[2];
		float t = dot_product(v0minOrig, N) / denominator;
		if (t < 0)
			continue;

		float P[3];
		P[0] = orig[0] + dir[0] * t;
		P[1] = orig[1] + dir[1] * t;
		P[2] = orig[2] + dir[2] * t;

		float C0[3], C1[3], C2[3];
		float V1minV0[3], V2minV1[3], V0minV2[3];
		float PminV0[3], PminV1[3], PminV2[3];

		V1minV0[0] = v1[0] - v0[0];	 V2minV1[0] = v2[0] - v1[0];  V0minV2[0] = v0[0] - v2[0];
		V1minV0[1] = v1[1] - v0[1];	 V2minV1[1] = v2[1] - v1[1];  V0minV2[1] = v0[1] - v2[1];
		V1minV0[2] = v1[2] - v0[2];	 V2minV1[2] = v2[2] - v1[2];  V0minV2[2] = v0[2] - v2[2];

		PminV0[0] = P[0] - v0[0];  PminV1[0] = P[0] - v1[0];  PminV2[0] = P[0] - v2[0];
		PminV0[1] = P[1] - v0[1];  PminV1[1] = P[1] - v1[1];  PminV2[1] = P[1] - v2[1];
		PminV0[2] = P[2] - v0[2];  PminV1[2] = P[2] - v1[2];  PminV2[2] = P[2] - v2[2];

		cross_product(V1minV0, PminV0, C0);
		cross_product(V2minV1, PminV1, C1);
		cross_product(V0minV2, PminV2, C2);

		if (dot_product(N, C0) < 0.0 ||
			dot_product(N, C1) < 0.0 ||
			dot_product(N, C2) < 0.0)
			continue;

		//float currentLength = distance(orig, P);
		if (t < minLength)
		{
			minLength = t;
			outLengthsData[tid] = minLength;
			outPointsData[tid * 3 + 0] = P[0];
			outPointsData[tid * 3 + 1] = P[1];
			outPointsData[tid * 3 + 2] = P[2];
		}
	}
}

#if 0
void calculateIntersection(const std::vector<TriangleData> &td,
						   const std::vector<vector3f> &dir,
						   const vector3f &orig,
						   std::vector<vector3f> &points,
						   std::vector<float> &lengths,
						   const unsigned int gridSizeBeta,
						   const unsigned int gridSizeEpsilon)
{
	QElapsedTimer tmr;
	tmr.start();

	IntersectionWizard& iw = IntersectionWizard::getInstance();

	/*������ �����*/
	if (objData == nullptr)
	{
		objDataBytes = td.size() * sizeof(TriangleData);
		objDataSize = td.size() * sizeof(TriangleData) / sizeof(float);
		objData = (float*)&td[0];// new float[objDataSize];
		memcpy(objData, &td[0], objDataBytes);
		HANDLE_ERROR(hipMalloc((void**)&dev_objData, objDataBytes));
		HANDLE_ERROR(hipMemcpy(dev_objData, objData, objDataBytes, hipMemcpyHostToDevice));
	}

	/*������ �������� �����������*/
	unsigned int dirDataBytes  = dir.size() * sizeof(vector3f);
	unsigned int dirDataSize = dir.size() * sizeof(vector3f) / sizeof(float);
	float *dirData = new float[dirDataSize];
	memcpy(dirData, &dir[0], dirDataBytes);
	float* dev_dirData;
	HANDLE_ERROR(hipMalloc((void**)&dev_dirData, dirDataBytes));
	HANDLE_ERROR(hipMemcpy(dev_dirData, dirData, dirDataBytes, hipMemcpyHostToDevice));

	/*������ �������*/
	unsigned int origDataBytes = sizeof(vector3f);
	unsigned int origDataSize = sizeof(vector3f) / sizeof(float);
	float *origData = new float[origDataSize];
	memcpy(origData, &orig, origDataBytes);
	float* dev_origData;
	HANDLE_ERROR(hipMalloc((void**)&dev_origData, origDataBytes));
	HANDLE_ERROR(hipMemcpy(dev_origData, origData, origDataBytes, hipMemcpyHostToDevice));

	/*�������� ������ ����� �����������*/
	unsigned int outPointsDataBytes = dir.size() * sizeof(vector3f);
	unsigned int outPointsDataSize = dir.size() * sizeof(vector3f) / sizeof(float);
	float *outPointsData  = new float[outPointsDataSize];
	float* dev_outPointsData;
	HANDLE_ERROR(hipHostMalloc((void**)&dev_outPointsData, outPointsDataBytes));

	/*�������� ������ ���������� �� ����� �����������*/
	unsigned int outLengthsDataBytes = dir.size() * sizeof(float);
	unsigned int outLengthsDataSize = dir.size();
	float *outLengthsData = new float[outLengthsDataSize];
	float* dev_outLengthsData;
	HANDLE_ERROR(hipHostMalloc((void**)&dev_outLengthsData, outLengthsDataBytes));

	dim3 grid(gridSizeBeta * gridSizeEpsilon, 1);
	calcIntersection<<<gridSizeBeta, gridSizeEpsilon >>>(dev_objData, dev_dirData, 
														 dev_origData, dev_outPointsData,
														 dev_outLengthsData, objDataSize);

	HANDLE_ERROR(hipMemcpy(outPointsData, dev_outPointsData, outPointsDataBytes, hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(outLengthsData, dev_outLengthsData, outLengthsDataBytes, hipMemcpyDeviceToHost));
	
	for (unsigned int i = 0; i < dir.size(); i++)
	{
		vector3f point(outPointsData[i * 3 + 0], outPointsData[i * 3 + 1], outPointsData[i * 3 + 2]);
		points.push_back(point);
		lengths.push_back(outLengthsData[i]);
	}

	hipFree(dev_dirData);
	hipFree(dev_origData);
	hipHostFree(dev_outPointsData);
	hipHostFree(dev_outLengthsData);

	delete[] dirData;
	delete[] origData;
	delete[] outPointsData;
	delete[] outLengthsData;

	quint64 time = tmr.elapsed();
	qDebug() << "Time = " << time;
}
#else
void calculateIntersection(const std::vector<TriangleData> &td,
	const std::vector<vector3f> &dir,
	const vector3f &orig,
	std::vector<vector3f> &points,
	std::vector<float> &lengths,
	const unsigned int gridSizeBeta,
	const unsigned int gridSizeEpsilon)
{
	QElapsedTimer tmr;
	tmr.start();

	IntersectionWizard& iw = IntersectionWizard::getInstance();

	/*������ �����*/
	float* dev_objData;
	HANDLE_ERROR(hipMalloc((void**)&dev_objData, iw.objData().bytes));
	HANDLE_ERROR(hipMemcpy(dev_objData, iw.objData().data, iw.objData().bytes, hipMemcpyHostToDevice));

	/*������ �������� �����������*/
	float* dev_dirData;
	HANDLE_ERROR(hipMalloc((void**)&dev_dirData, iw.dirData().bytes));
	HANDLE_ERROR(hipMemcpy(dev_dirData, iw.dirData().data, iw.dirData().bytes, hipMemcpyHostToDevice));

	/*������ �������*/
	float* dev_origData;
	HANDLE_ERROR(hipMalloc((void**)&dev_origData, iw.origData().bytes));
	HANDLE_ERROR(hipMemcpy(dev_origData, iw.origData().data, iw.origData().bytes, hipMemcpyHostToDevice));

	/*�������� ������ ����� �����������*/
	unsigned int outPointsDataBytes = dir.size() * sizeof(vector3f);
	unsigned int outPointsDataSize = dir.size() * sizeof(vector3f) / sizeof(float);
	float *outPointsData = new float[outPointsDataSize];
	float* dev_outPointsData;
	HANDLE_ERROR(hipHostMalloc((void**)&dev_outPointsData, outPointsDataBytes));

	/*�������� ������ ���������� �� ����� �����������*/
	unsigned int outLengthsDataBytes = dir.size() * sizeof(float);
	unsigned int outLengthsDataSize = dir.size();
	float *outLengthsData = new float[outLengthsDataSize];
	float* dev_outLengthsData;
	HANDLE_ERROR(hipHostMalloc((void**)&dev_outLengthsData, outLengthsDataBytes));

	dim3 grid(gridSizeBeta * gridSizeEpsilon, 1);
	calcIntersection << <gridSizeBeta, gridSizeEpsilon >> > (dev_objData, dev_dirData,
		dev_origData, dev_outPointsData,
		dev_outLengthsData, iw.objData().size);

	HANDLE_ERROR(hipMemcpy(outPointsData, dev_outPointsData, outPointsDataBytes, hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(outLengthsData, dev_outLengthsData, outLengthsDataBytes, hipMemcpyDeviceToHost));

	for (unsigned int i = 0; i < dir.size(); i++)
	{
		vector3f point(outPointsData[i * 3 + 0], outPointsData[i * 3 + 1], outPointsData[i * 3 + 2]);
		points.push_back(point);
		lengths.push_back(outLengthsData[i]);
	}

	hipFree(dev_objData);
	hipFree(dev_dirData);
	hipFree(dev_origData);
	hipHostFree(dev_outPointsData);
	hipHostFree(dev_outLengthsData);

	delete[] outPointsData;
	delete[] outLengthsData;

	quint64 time = tmr.elapsed();
	qDebug() << "Time = " << time;
}

#endif
