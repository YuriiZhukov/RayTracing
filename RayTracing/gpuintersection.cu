#include "hip/hip_runtime.h"
#include <qdebug.h>
#include <qelapsedtimer.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "structs.h"
#include "intersectionwizard.h"

void startTimer();
void stopTimer(float& dest);

hipEvent_t start, stop;

static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__device__ float dot_product(float *a, float *b)
{
	return (a[0] * b[0] + a[1] * b[1] + a[2] * b[2]);
}

__device__ void cross_product(float *a, float *b, float *dest)
{
	dest[0] = a[1] * b[2] - a[2] * b[1];
	dest[1] = a[2] * b[0] - a[0] * b[2];
	dest[2] = a[0] * b[1] - a[1] * b[0];
}

__global__ void calcIntersection(float *objData, float *dirData, float *origData, float *outPointsData, float *outLengthsData,
	unsigned int objDataSize, unsigned int raysCount)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x; //ID ��������� �� ���� ��� (���� �����������) �� *dirData
	if (tid > raysCount)
		return;

	while (tid < raysCount)
	{
		/*������ � dirData:
		0 = float x, 1 = float y, 2 = float z
		3 = float x, 4 = float y, 5 = float z � �.� �� (dirDataSize / sizeof(float)) */

		float dir[3];
		dir[0] = dirData[tid * 3 + 0];
		dir[1] = dirData[tid * 3 + 1];
		dir[2] = dirData[tid * 3 + 2];

		float orig[3];
		orig[0] = origData[0];
		orig[1] = origData[1];
		orig[2] = origData[2];

		float outPoint[3];
		outPoint[0] = 0;
		outPoint[1] = 0;
		outPoint[2] = 0;

		float minLength = 10000.0;
		
		/*������ � objData:
		0, 1, 2 - ������� 1� �����, 3, 4, 5 - ���������� 1� �����
		6, 7, 8 - ������� 2� �����, 9, 10, 11 - ���������� 2� �����
		12, 13, 14 - ������� 3� �����, 15, 16, 17 - ���������� 3� �����
		� �.� �� (objDataSize / sizeof(float)) */
		for (int i = 0; i < objDataSize; i += 18) //18 - ������ ������ TriangleData
		{
			float epsilon = 0.000001;
			float N[3], v0[3], v1[3], v2[3];

			N[0] = objData[i + 0];
			N[1] = objData[i + 1];
			N[2] = objData[i + 2];

			v0[0] = objData[i + 3];  v1[0] = objData[i + 9];   v2[0] = objData[i + 15];
			v0[1] = objData[i + 4];  v1[1] = objData[i + 10];  v2[1] = objData[i + 16];
			v0[2] = objData[i + 5];  v1[2] = objData[i + 11];  v2[2] = objData[i + 17];

			float denominator = dot_product(dir, N);
			if ((denominator >= 0.0 && denominator < epsilon) ||
				(denominator <  0.0 && denominator > -epsilon))
				continue;

			float v0minOrig[3];
			v0minOrig[0] = v0[0] - orig[0];
			v0minOrig[1] = v0[1] - orig[1];
			v0minOrig[2] = v0[2] - orig[2];
			float t = dot_product(v0minOrig, N) / denominator;
			if (t < 0)
				continue;

			float P[3];
			P[0] = orig[0] + dir[0] * t;
			P[1] = orig[1] + dir[1] * t;
			P[2] = orig[2] + dir[2] * t;

			float C0[3], C1[3], C2[3];
			float V1minV0[3], V2minV1[3], V0minV2[3];
			float PminV0[3], PminV1[3], PminV2[3];

			V1minV0[0] = v1[0] - v0[0];	 V2minV1[0] = v2[0] - v1[0];  V0minV2[0] = v0[0] - v2[0];
			V1minV0[1] = v1[1] - v0[1];	 V2minV1[1] = v2[1] - v1[1];  V0minV2[1] = v0[1] - v2[1];
			V1minV0[2] = v1[2] - v0[2];	 V2minV1[2] = v2[2] - v1[2];  V0minV2[2] = v0[2] - v2[2];

			PminV0[0] = P[0] - v0[0];  PminV1[0] = P[0] - v1[0];  PminV2[0] = P[0] - v2[0];
			PminV0[1] = P[1] - v0[1];  PminV1[1] = P[1] - v1[1];  PminV2[1] = P[1] - v2[1];
			PminV0[2] = P[2] - v0[2];  PminV1[2] = P[2] - v1[2];  PminV2[2] = P[2] - v2[2];

			cross_product(V1minV0, PminV0, C0);
			cross_product(V2minV1, PminV1, C1);
			cross_product(V0minV2, PminV2, C2);

			if (dot_product(N, C0) < 0.0 ||
				dot_product(N, C1) < 0.0 ||
				dot_product(N, C2) < 0.0)
				continue;

			//float currentLength = distance(orig, P);
			if (t < minLength)
			{
				minLength = t;
				outPoint[0] = P[0];
				outPoint[1] = P[1];
				outPoint[2] = P[2];
			}
		}

		outLengthsData[tid] = minLength;
		outPointsData[tid * 3 + 0] = outPoint[0];
		outPointsData[tid * 3 + 1] = outPoint[1];
		outPointsData[tid * 3 + 2] = outPoint[2];

		tid += gridDim.x * blockDim.x;
	}
}

void calculateIntersection(std::vector<vector3f> &points, std::vector<float> &lengths)
{
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	float timeToMalloc;
	float timeToCalculate;
	float timeToMemcpy;
	float timeToFillVectors;
	float timeToFreeMemory;

	IntersectionWizard& iw = IntersectionWizard::getInstance();
	if (!iw.dataCorrect())
	{
		qDebug() << "Data incorrect";
		return;
	}
	
startTimer();
	/*������ �����*/
	float* dev_objData;
	HANDLE_ERROR(hipMalloc((void**)&dev_objData, iw.objData().bytes));
	HANDLE_ERROR(hipMemcpy(dev_objData, iw.objData().data, iw.objData().bytes, hipMemcpyHostToDevice));

	/*������ �������� �����������*/
	float* dev_dirData;
	HANDLE_ERROR(hipMalloc((void**)&dev_dirData, iw.dirData().bytes));
	HANDLE_ERROR(hipMemcpy(dev_dirData, iw.dirData().data, iw.dirData().bytes, hipMemcpyHostToDevice));

	/*������ �������*/
	float* dev_origData;
	HANDLE_ERROR(hipMalloc((void**)&dev_origData, iw.origData().bytes));
	HANDLE_ERROR(hipMemcpy(dev_origData, iw.origData().data, iw.origData().bytes, hipMemcpyHostToDevice));

	/*�������� ������ ����� �����������*/
	unsigned int outPointsDataBytes = iw.dirData().bytes;
	unsigned int outPointsDataSize = iw.dirData().size;
	float *outPointsData = new float[outPointsDataSize];
	float* dev_outPointsData;
	HANDLE_ERROR(hipMalloc((void**)&dev_outPointsData, outPointsDataBytes));

	/*�������� ������ ���������� �� ����� �����������*/
	unsigned int outLengthsDataBytes = iw.dirData().count * sizeof(float);
	unsigned int outLengthsDataSize = iw.dirData().count;
	float *outLengthsData = new float[outLengthsDataSize];
	float* dev_outLengthsData;
	HANDLE_ERROR(hipMalloc((void**)&dev_outLengthsData, outLengthsDataBytes));	

stopTimer(timeToMalloc);

startTimer();
	calcIntersection << <iw.dirData().count / 512 + 1,512 >> >
													(dev_objData, dev_dirData, dev_origData, 
													 dev_outPointsData, dev_outLengthsData,
													 iw.objData().size, iw.dirData().count);
stopTimer(timeToCalculate);

startTimer();
	HANDLE_ERROR(hipMemcpy(outPointsData, dev_outPointsData, outPointsDataBytes, hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(outLengthsData, dev_outLengthsData, outLengthsDataBytes, hipMemcpyDeviceToHost));
stopTimer(timeToMemcpy);

	lengths.resize(iw.dirData().count);
	points.resize(iw.dirData().count);

	for (unsigned int i = 0; i < iw.dirData().count; i++)
	{
		vector3f point(outPointsData[i * 3 + 0], outPointsData[i * 3 + 1], outPointsData[i * 3 + 2]);
		points[i] = point;
		lengths[i] = outLengthsData[i];
	}

startTimer();
	hipFree(dev_objData);
	hipFree(dev_dirData);
	hipFree(dev_origData);
	hipFree(dev_outPointsData);
	hipFree(dev_outLengthsData);
stopTimer(timeToFreeMemory);

	delete[] outPointsData;
	delete[] outLengthsData;

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	qDebug() << "Malloc ms    = " << timeToMalloc;
	qDebug() << "Calculate ms = " << timeToCalculate;
	qDebug() << "Memcpy ms    = " << timeToMemcpy;;
	qDebug() << "Free mem ms  = " << timeToFreeMemory << "\n";
}

void startTimer()
{
	HANDLE_ERROR(hipEventRecord(start, 0));
}

void stopTimer(float& dest)
{
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&dest, start, stop));
}